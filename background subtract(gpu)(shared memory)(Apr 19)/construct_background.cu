#include "hip/hip_runtime.h"
#include "background_subtraction.h"
#include ""

__global__ void construct_background_shared_mem(uchar3 *img_1, uchar3 *img_2, uchar3 *img_3, uchar3 *img_4,
	uchar3 *img_5, uchar3 *img, int img_width, int img_height) {

	__shared__ uchar3 ds_img_1[TILE_WIDTH][TILE_WIDTH];
	__shared__ uchar3 ds_img_2[TILE_WIDTH][TILE_WIDTH];
	__shared__ uchar3 ds_img_3[TILE_WIDTH][TILE_WIDTH];
	__shared__ uchar3 ds_img_4[TILE_WIDTH][TILE_WIDTH];
	__shared__ uchar3 ds_img_5[TILE_WIDTH][TILE_WIDTH];

	int tx, ty, img_x, img_y;

	// Make sure the tile is within the padded image
	int block_end_col = blockIdx.x*blockDim.x + blockDim.x;
	int block_end_row = blockIdx.y*blockDim.y + blockDim.y;
	const unsigned int blockEndClampedCol = ((block_end_col<img_width) ? block_end_col : img_width);
	const unsigned int blockEndClampedRow = ((block_end_row<img_height) ? block_end_row : img_height);

	// Load data to tiles
	tx = threadIdx.x;
	ty = threadIdx.y;
	img_x = blockIdx.x*blockDim.x + threadIdx.x;
	img_y = blockIdx.y*blockDim.y + threadIdx.y;
	if ((img_y < blockEndClampedRow) && (img_x < blockEndClampedCol)) {
		ds_img_1[ty][tx] = img_1[img_y*img_width + img_x];
		ds_img_2[ty][tx] = img_2[img_y*img_width + img_x];
		ds_img_3[ty][tx] = img_3[img_y*img_width + img_x];
		ds_img_4[ty][tx] = img_4[img_y*img_width + img_x];
		ds_img_5[ty][tx] = img_5[img_y*img_width + img_x];
	}

	// Barrier synchronization
	__syncthreads();

	// Construct the background by averaging pixels within 5 images
	tx = threadIdx.x;
	ty = threadIdx.y;
	img_x = blockIdx.x*blockDim.x + threadIdx.x;
	img_y = blockIdx.y*blockDim.y + threadIdx.y;
	if ((img_y>=(blockIdx.y*blockDim.y)) && (img_y<blockEndClampedRow) && (img_x>=(blockIdx.x*blockDim.x)) && (img_x < blockEndClampedCol)) {

		img[img_y*img_width + img_x].x 
			= unsigned char((0.2) * (ds_img_1[ty][tx].x + ds_img_2[ty][tx].x + ds_img_3[ty][tx].x + ds_img_4[ty][tx].x + ds_img_5[ty][tx].x));
		img[img_y*img_width + img_x].y
			= unsigned char((0.2) * (ds_img_1[ty][tx].y + ds_img_2[ty][tx].y + ds_img_3[ty][tx].y + ds_img_4[ty][tx].y + ds_img_5[ty][tx].y));
		img[img_y*img_width + img_x].z
			= unsigned char((0.2) * (ds_img_1[ty][tx].z + ds_img_2[ty][tx].z + ds_img_3[ty][tx].z + ds_img_4[ty][tx].z + ds_img_5[ty][tx].z));
	}

}

// The function for constructing background
void construct_background(uchar3 *d_bg_1, uchar3 *d_bg_2, uchar3 *d_bg_3,
	uchar3 *d_bg_4, uchar3 *d_bg_5, uchar3 *d_bg, int img_width, int img_height) {

	dim3 gridSize(int(ceil(img_width / BLOCK) + 1), int(ceil(img_height / BLOCK) + 1), 1);
	dim3 blockSize(int(BLOCK), int(BLOCK), 1);

	construct_background_shared_mem KERNEL_ARGS2(gridSize, blockSize) (d_bg_1, d_bg_2, d_bg_3, d_bg_4, d_bg_5, d_bg, img_width, img_height);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
}